
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

// Parameters
#define TILE_N 64     // tile dimension
#define THREAD_N 16   // number of threads for each
#define TRIALS_N 5000 // trials to run and average over

__global__ void transpose(float *a, float *b);
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true);
inline void fillMatrix(float *m, size_t n);
inline void printMatrix(float *m, size_t n);

__global__ void transpose(float *a, float *b) {
    /* Calculate global tile indices for the thread w.r.t the input matrix */
    int x = blockIdx.x * TILE_N + threadIdx.x;
    int y = blockIdx.y * TILE_N + threadIdx.y;
    int width = gridDim.x * TILE_N;
    
    /* Load data into the tiles */
    __shared__ float tile[TILE_N][TILE_N+1];
    for (int j = 0; j < TILE_N; j += THREAD_N)
        tile[threadIdx.y+j][threadIdx.x] = a[(y+j)*width + x];
    
    /* Wait for all threads to finish */
    __syncthreads();

    /* Calculate global tile indices for the thread w.r.t the output matrix */
    x = blockIdx.y * TILE_N + threadIdx.x;
    y = blockIdx.x * TILE_N + threadIdx.y;

    /* Load data into the output matrix */
    for (int j = 0; j < TILE_N; j += THREAD_N)
        b[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(int argc, char **argv) {
    /* Check that the necessary parameter is passed */
    if (argc < 2) {
        fprintf(stderr, "Error: Missing size.\n");
        fprintf(stderr, "Usage: transpose <n> [OPTIONAL: <device>]\n");
        return -1;
    }

    /* Parse parameters */
    size_t N = 1 << atoi(argv[1]);
    int device = (argc > 2) ? atoi(argv[2]) : 0;

    std::cout << "N = " << N << std::endl;

    /* Allocate the required host memory */
    float *a = new float[N*N];
    float *b = new float[N*N];
    fillMatrix(a, N); // fill with test data
    
    /* Allocate the required device memory */
    float *d_a, *d_b;
    gpuErrchk(hipMalloc((void**)&d_a, (N * N) * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_b, (N * N) * sizeof(float)));

    /* Copy the matrix into device memory */
    gpuErrchk(hipMemcpy(d_a, a, (N * N) * sizeof(float), hipMemcpyHostToDevice));

    /* Prepare timing events */
    hipEvent_t start, stop;
    float time;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    /* Calculate number of blocks and threads per block */
    dim3 gridDim(N / TILE_N, N / TILE_N);
    dim3 blockDim(TILE_N, THREAD_N);

    /* Run the kernel many times to get a statistically significant amount of data */
    gpuErrchk(hipEventRecord(start, 0)); // start timer
    for (int i = 0; i < TRIALS_N; i++)
        transpose<<<gridDim, blockDim>>>(d_a, d_b);
    
    /* Copy back the results */
    gpuErrchk(hipMemcpy(b, d_b, (N * N) * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipPeekAtLastError());

    /* Stop timer and estimate bandwidth */
    gpuErrchk(hipEventRecord(stop, 0));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&time, start, stop));
    size_t bytes = 2 * 4 * N * N * TRIALS_N;
    double gb = bytes / 1024. / 1024. / 1024.;
    double bandwidth = gb / (time / 1000.);
    std::cout << "Bandwidth: " << bandwidth << " GB/s" << std::endl;

#ifdef DEBUG
    /* Print before and after */
    std::cout << "A: " << std::endl;
    printMatrix(a, N);

    std::cout << "B: " << std::endl;
    printMatrix(b, N);
#endif

    /* Free up resources */
    hipFree(d_a);
    hipFree(d_b);
    delete []a;
    delete []b;

    return 0;
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void fillMatrix(float *m, size_t n) {
    for (size_t i = 0; i < n; i++)
        for (size_t j = 0; j < n; j++)
            m[(i*n)+j] = j*i+1 / (float)(i+1);
}

void printMatrix(float *m, size_t n) {
    std::cout.setf(std::ios::fixed);
    std::cout.setf(std::ios::showpoint);
    std::cout.precision(2);
    for (size_t j = 0; j < n; j++) {
        for (size_t i = 0; i < n; i++) {
            std::cout << std::setw(8) << m[(j*n) + i] << " ";
        }
        std::cout << std::endl;
    }
}